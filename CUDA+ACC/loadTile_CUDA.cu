/* Cuda Code for Game Of Life 
*  Priscilla Kelly June 21, 2016
*/


#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"

/***************************************/
/* External c subroutine for CUDA      */
/***************************************/
extern "C" void call_loadTile_CUDA(int flag, int elements, int *Matrix, int **pointer2device) {

	size_t matSize = elements*sizeof(int);
	hipError_t err = hipSuccess;

	if (flag == 0) {

		/***************************************/
		/* Allocate Matrix to the GPU          */
		/***************************************/
		int *device;
		err = hipMalloc(&device, matSize);
		if(err != hipSuccess) {
			fprintf(stderr, "Failed to allocate device vector (error code %s)!\n",
					hipGetErrorString(err));
			exit(EXIT_FAILURE);		
		}
		// move matrix to device
		err = hipMemcpy(device,Matrix,matSize,hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			fprintf(stderr, "%s Failed at line %s !\n",__FILE__,__LINE__);
			exit(EXIT_FAILURE);
		}
        
		*pointer2device = device;
		return;
	}

	if (flag == 1) {

		/***************************************/
		/* Free Device Global Memory           */
		/***************************************/
		err = hipFree(*pointer2device);
		if (err != hipSuccess){
			fprintf(stderr, "Failed to free device!\n");
			exit(EXIT_FAILURE);
		}
	}
	
	if (flag == 3) {
		hipDeviceSynchronize();
		int *host_subMat = (int *)malloc(matSize);	
		if(host_subMat == NULL) {
			fprintf(stderr, "Failed to alocate host vector!\n");
 			exit(EXIT_FAILURE);     
		}
		err = hipMemcpy(host_subMat,*pointer2device,matSize,hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			fprintf(stderr,"Failed to copy the submat from device (error code %s)!\n",hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		int i,j;
		int c=0;
		printf("New Rank\n");
		for(i=0;i<6;i++) {
			printf("[");
			for(j=0;j<6;j++) {
				printf(" %d ",host_subMat[c]);
				c++;	
			}	
			printf("]\n");
		}
		printf("\n");
		return;
	}	
}	
